#include "hip/hip_runtime.h"
// SPDX-FileCopyrightText: Copyright (c) 2024, NVIDIA CORPORATION. All rights reserved.Add commentMore actions
// SPDX-License-Identifier: BSD-3-Clause

// Because CUB cannot inspect the transformation function, we cannot add any tunings based on the results of this
// benchmark. Its main use is to detect regressions.

// %RANGE% TUNE_THREADS tpb 128:1024:128
// %RANGE% TUNE_ALGORITHM alg 0:1:1

#include "common.h"

// This benchmark tests overlapping memory regions for reading and is compute intensive

inline constexpr auto startA = 0.1;
inline constexpr auto startB = 0.2;

static void vectoradd(nvbench::state& state)
{
  const auto n = narrow<int32_t>(state.get_int64("Elements{io}"));
  thrust::device_vector<__half> in1(n, startA);
  thrust::device_vector<__half> in2(n, startB);
  thrust::device_vector<__half> out(n);

  state.add_element_count(n);
  state.add_global_memory_reads<__half>(n);
  state.add_global_memory_reads<__half>(n);
  state.add_global_memory_writes<__half>(n);

  bench_transform(state, ::cuda::std::tuple{in1.begin(), in2.begin()}, out.begin(), n, ::cuda::std::plus<>{});
}

NVBENCH_BENCH(vectoradd).set_name("vectoradd").add_int64_power_of_two_axis("Elements{io}", nvbench::range(16, 28, 4));
