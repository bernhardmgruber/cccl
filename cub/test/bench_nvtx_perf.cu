#include "hip/hip_runtime.h"
// Tests the overhead of the NVTX API when enabled. Compile and run from this directory with:
//   nvcc bench_nvtx_perf.cu -I../../cub -I../../thrust -I../../libcudacxx/include && ./a.out
// Variations:
// * Add `-DNVTX_DISABLE` to disable the NVTX API, should be faster
// * Run binary with nsys profile ./a.out to attach Nsight Systems , should be slower

#include <cub/device/device_for.cuh>

#include <thrust/iterator/counting_iterator.h>

#include <chrono>

struct Op
{
  _CCCL_HOST_DEVICE void operator()(int) const
  {
  }
};

constexpr auto reps = 10'000;

int main()
{
  using clock = std::chrono::high_resolution_clock;

  thrust::counting_iterator<int> it{0};

  const auto start = clock::now();
  for(int i = 0; i < reps; i++) {
    hipcub::DeviceFor::ForEach(it, it + 1, Op{});
  }
  const auto end = clock::now();
  std::cout << "Average launch time: " << std::chrono::duration<double, std::micro>(end - start).count() / reps << "μs\n";

  hipDeviceSynchronize();
}
